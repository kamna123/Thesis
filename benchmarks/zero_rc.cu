#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
int main(int argc, char** argv)
{
   
    int q[20],i;
__global__ void _AFFINE_KERNEL_0(int* ,int ,int ,int ,int );


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_0, end_0, mid_start_0, mid_end_0;
		double runTime_0, pre_time_0, post_time_0, computeTime_0;
	#endif
	int _SZ_q_0 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_0);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_q_0;
	hipMalloc((void**) &_DEV_q_0,sizeof(int) *_SZ_q_0);
	hipMemcpy(_DEV_q_0,q,sizeof(int)*_SZ_q_0, hipMemcpyHostToDevice);
	int _NUM_THREADS_0= 20;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_0);
	#endif

	int _CUDA_TILE_0;
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>( _DEV_q_0, _SZ_q_0,0,18, _CUDA_TILE_0);
		hipDeviceSynchronize();
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_0);
	#endif
	hipMemcpy(q, _DEV_q_0,sizeof(int) *_SZ_q_0, hipMemcpyDeviceToHost);
	hipFree(_DEV_q_0);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_0);

		pre_time_0 = (double) ((((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
		post_time_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec)) / 1000000000;
		computeTime_0 = (double) ((((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec) - (((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec)) / 1000000000;
		runTime_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%%d,%%d,%%d,%%d,%%d,%%.14f,%%.14f,%%.14f,%%.14f,%%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time_0,computeTime_0,post_time_0,runTime_0,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%%f\n",runTime_0);
	#endif
	#endif
   
__global__ void _AFFINE_KERNEL_1(int* ,int ,int ,int ,int ,int ,int );


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_1, end_1, mid_start_1, mid_end_1;
		double runTime_1, pre_time_1, post_time_1, computeTime_1;
	#endif
	int _SZ_q_1 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_1);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int*_DEV_q_1;
	hipMalloc((void**) &_DEV_q_1,sizeof(int) *_SZ_q_1);
	hipMemcpy(_DEV_q_1,q,sizeof(int) *_SZ_q_1, hipMemcpyHostToDevice);
	int _NUM_THREADS_1= 20;
	float _NUM_BLOCKS_1=1;
	int _NUM_TILE_1=1;
	dim3 _THREADS_1(512);
	dim3 _BLOCKS_1(1);
	if(_NUM_THREADS_1 < _NTHREAD)
	{
		_THREADS_1.x=_NUM_THREADS_1;
	}
	else {
		 _THREADS_1.x=_NTHREAD;
		_NUM_BLOCKS_1=(_NUM_THREADS_1 % _NTHREAD == 0)?(_NUM_THREADS_1/_NTHREAD):((_NUM_THREADS_1/_NTHREAD)+1);
		if(_NUM_BLOCKS_1<_NBLOCK)
			_BLOCKS_1.x=_NUM_BLOCKS_1;
		else {
			_BLOCKS_1.x=_NBLOCK;
			int temp_1=_NUM_BLOCKS_1;
			_NUM_TILE_1=(temp_1 % _NBLOCK == 0)?(_NUM_BLOCKS_1/_NBLOCK):((_NUM_BLOCKS_1/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_1);
	#endif

	int _CUDA_TILE_1;
	for(i=0;i<18;i+=2)
	for(_CUDA_TILE_1=0;_CUDA_TILE_1<_NUM_TILE_1;_CUDA_TILE_1++)
	{		_AFFINE_KERNEL_1<<<_BLOCKS_1,_THREADS_1>>>( _DEV_q_1, _SZ_q_1,1,i,0,18,_CUDA_TILE_1);
		hipDeviceSynchronize();
	}
	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_1);
	#endif
	hipMemcpy(q, _DEV_q_1,sizeof(int) *_SZ_q_1, hipMemcpyDeviceToHost);
	hipFree(_DEV_q_1);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_1);
		pre_time_1 = (double) ((((&mid_start_1)->tv_sec * 1000000000) + (&mid_start_1)->tv_nsec) - (((&start_1)->tv_sec * 1000000000) + (&start_1)->tv_nsec)) / 1000000000;
		post_time_1 = (double) ((((&end_1)->tv_sec * 1000000000) + (&end_1)->tv_nsec) - (((&mid_end_1)->tv_sec * 1000000000) + (&mid_end_1)->tv_nsec)) / 1000000000;
		computeTime_1 = (double) ((((&mid_end_1)->tv_sec * 1000000000) + (&mid_end_1)->tv_nsec) - (((&mid_start_1)->tv_sec * 1000000000) + (&mid_start_1)->tv_nsec)) / 1000000000;
		runTime_1 = (double) ((((&end_1)->tv_sec * 1000000000) + (&end_1)->tv_nsec) - (((&start_1)->tv_sec * 1000000000) + (&start_1)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%%d,%%d,%%d,%%d,%%d,%%.14f,%%.14f,%%.14f,%%.14f,%%d\n",N,_NTHREAD*_NBLOCK,_THREADS_1.x,_BLOCKS_1.x,data,pre_time_1,computeTime_1,post_time_1,runTime_1,_CUDA_TILE_1);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%%f\n",runTime_1);
	#endif
	#endif
    
	
    return 0;
}




__global__ void _AFFINE_KERNEL_0(int* q,int  _SZ_q_0,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
q[1+i] =(2 *(i + 1));
}}



__global__ void _AFFINE_KERNEL_1(int* q,int  _SZ_q_1,int phi_count, int CUDA_i,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_i<=i)&&(i<(CUDA_i+2))&&(i<CUDA_U_i)){
q[19-i] = q[19-i-2];
}}

