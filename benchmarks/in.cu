#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
/*cb gf*/
//int gcdf()
int main()
{
     int x[20],y[20],z[10],f[20];
   int i,a[50],k=5,j,b[300],pi=3.14,w[200];
 
 
   //fasdf
k=5;

__global__ void _AFFINE_KERNEL_0(int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_0, end_0, mid_start_0, mid_end_0;
		double runTime_0, pre_time_0, post_time_0, computeTime_0;
	#endif
	int _SZ_a_0 = 50;
	int _SZ_b_0 = 300;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_0);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_a_0;
	hipMalloc((void**) &_DEV_a_0,sizeof(int) *_SZ_a_0);
	hipMemcpy(_DEV_a_0,a,sizeof(int) *_SZ_a_0, hipMemcpyHostToDevice);
	int *_DEV_b_0;
	hipMalloc((void**) &_DEV_b_0,sizeof(int) *_SZ_b_0);
	hipMemcpy(_DEV_b_0,b,sizeof(int) *_SZ_b_0, hipMemcpyHostToDevice);
	int _NUM_THREADS_0 = 50;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}
	int ID_1_0, ID_2_0, START[1];

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_0);
	#endif

	int _CUDA_TILE_0;
	int Phi_0[1]={0};
	int loopUpperLimits_0[1]={45};
	for(ID_1_0=1;ID_1_0<=45/0+1;ID_1_0++)
	{
		for(ID_2_0=0;ID_2_0<1;ID_2_0++)
		{
			if(Phi_0[ID_2_0]>=0)
				START[ID_2_0]=(ID_1_0-1)*Phi_0[ID_2_0];
			else
				START[ID_2_0]=loopUpperLimits_0[ID_2_0]+(ID_1_0-1)*Phi_0[ID_2_0];
		}
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>( _DEV_a_0, _SZ_a_0, _DEV_b_0, _SZ_b_0,START[0], MIN(START[0]+0, 45), _CUDA_TILE_0);
			hipDeviceSynchronize();
	}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_0);
	#endif
	hipMemcpy(a, _DEV_a_0,sizeof(int) *_SZ_a_0, hipMemcpyDeviceToHost);
	hipMemcpy(b, _DEV_b_0,sizeof(int) *_SZ_b_0, hipMemcpyDeviceToHost);
	hipFree(_DEV_a_0);
	hipFree(_DEV_b_0);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_0);
		pre_time_0 = (double) ((((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
		post_time_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec)) / 1000000000;
		computeTime_0 = (double) ((((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec) - (((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec)) / 1000000000;
		runTime_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%%d,%%d,%%d,%%d,%%d,%%.14f,%%.14f,%%.14f,%%.14f,%%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time_0,computeTime_0,post_time_0,runTime_0,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%%f\n",runTime_0);
	#endif
	#endif
  }



__global__ void _AFFINE_KERNEL_0(int* a,int  _SZ_a_0,int* b,int  _SZ_b_0,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
a[1+i+10] = a[1+i+2];
b[1+i+1] = a[1+i];
a[1+i] = b[1+i+1];
}}


