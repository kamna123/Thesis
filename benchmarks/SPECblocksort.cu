#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
int main()
{
    int block[20],quadrant[20],i,j,k;
    //for(i=1;i<=20;i=i+1)
     i=1;
     while(i<=20)
    {
        block[i]=2*(i+1);
        quadrant[i]=3*(i+1);
         i++;
    }
    // PARALLELIZE IN CUDA START
__global__ void _AFFINE_KERNEL_0(int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_0, end_0, mid_start_0, mid_end_0;
		double runTime_0, pre_time_0, post_time_0, computeTime_0;
	#endif
	int _SZ_block_0 = 20;
	int _SZ_quadrant_0 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_0);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_block_0;
	hipMalloc((void**) &_DEV_block_0,sizeof(int) *_SZ_block_0);
	hipMemcpy(_DEV_block_0,block,sizeof(int) *_SZ_block_0, hipMemcpyHostToDevice);
	int *_DEV_quadrant_0;
	hipMalloc((void**) &_DEV_quadrant_0,sizeof(int) *_SZ_quadrant_0);
	hipMemcpy(_DEV_quadrant_0,quadrant,sizeof(int) *_SZ_quadrant_0, hipMemcpyHostToDevice);
	int _NUM_THREADS_0 = 20;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}
	int ID_1_0, ID_2_0, START[1];

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_0);
	#endif

	int _CUDA_TILE_0;
	int Phi_0[1]={15};
	int loopUpperLimits_0[1]={19};
	for(ID_1_0=1;ID_1_0<=19/15+1;ID_1_0++)
	{
		for(ID_2_0=0;ID_2_0<1;ID_2_0++)
		{
			if(Phi_0[ID_2_0]>=0)
				START[ID_2_0]=(ID_1_0-1)*Phi_0[ID_2_0];
			else
				START[ID_2_0]=loopUpperLimits_0[ID_2_0]+(ID_1_0-1)*Phi_0[ID_2_0];
		}
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>( _DEV_block_0, _SZ_block_0, _DEV_quadrant_0, _SZ_quadrant_0,START[0], MIN(START[0]+15, 19), _CUDA_TILE_0);
			hipDeviceSynchronize();
	}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_0);
	#endif
	hipMemcpy(block, _DEV_block_0,sizeof(int) *_SZ_block_0, hipMemcpyDeviceToHost);
	hipMemcpy(quadrant, _DEV_quadrant_0,sizeof(int) *_SZ_quadrant_0, hipMemcpyDeviceToHost);
	hipFree(_DEV_block_0);
	hipFree(_DEV_quadrant_0);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_0);
		pre_time_0 = (double) ((((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
		post_time_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec)) / 1000000000;
		computeTime_0 = (double) ((((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec) - (((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec)) / 1000000000;
		runTime_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%%d,%%d,%%d,%%d,%%d,%%.14f,%%.14f,%%.14f,%%.14f,%%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time_0,computeTime_0,post_time_0,runTime_0,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%%f\n",runTime_0);
	#endif
	#endif
    // PARALLELIZE IN CUDA END
}




__global__ void _AFFINE_KERNEL_0(int* block,int  _SZ_block_0,int* quadrant,int  _SZ_quadrant_0,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
block[15+1+i] = block[1+i];
quadrant[15+1+i] = 0;
}}


