#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
int main()
{
    int x[20];
    int w[20],i,j,k;
__global__ void _AFFINE_KERNEL_0(int* ,int ,int* ,int ,int ,int ,int );


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_0, end_0, mid_start_0, mid_end_0;
		double runTime_0, pre_time_0, post_time_0, computeTime_0;
	#endif
	int _SZ_w_0 = 20;
	int _SZ_x_0 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_0);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_w_0;
	hipMalloc((void**) &_DEV_w_0,sizeof(int) *_SZ_w_0);
	hipMemcpy(_DEV_w_0,w,sizeof(int)*_SZ_w_0, hipMemcpyHostToDevice);
	int *_DEV_x_0;
	hipMalloc((void**) &_DEV_x_0,sizeof(int) *_SZ_x_0);
	hipMemcpy(_DEV_x_0,x,sizeof(int)*_SZ_x_0, hipMemcpyHostToDevice);
	int _NUM_THREADS_0= 20;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_0);
	#endif

	int _CUDA_TILE_0;
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>( _DEV_w_0, _SZ_w_0, _DEV_x_0, _SZ_x_0,0,9, _CUDA_TILE_0);
		hipDeviceSynchronize();
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_0);
	#endif
	hipMemcpy(w, _DEV_w_0,sizeof(int) *_SZ_w_0, hipMemcpyDeviceToHost);
	hipMemcpy(x, _DEV_x_0,sizeof(int) *_SZ_x_0, hipMemcpyDeviceToHost);
	hipFree(_DEV_w_0);
	hipFree(_DEV_x_0);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_0);

		pre_time_0 = (double) ((((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
		post_time_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec)) / 1000000000;
		computeTime_0 = (double) ((((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec) - (((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec)) / 1000000000;
		runTime_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%%d,%%d,%%d,%%d,%%d,%%.14f,%%.14f,%%.14f,%%.14f,%%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time_0,computeTime_0,post_time_0,runTime_0,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%%f\n",runTime_0);
	#endif
	#endif
    // PARALLELIZE IN CUDA START
__global__ void _AFFINE_KERNEL_1(int* ,int ,int* ,int ,int ,int ,int ,int ,int );


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_1, end_1, mid_start_1, mid_end_1;
		double runTime_1, pre_time_1, post_time_1, computeTime_1;
	#endif
	int _SZ_w_1 = 20;
	int _SZ_x_1 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_1);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int*_DEV_w_1;
	hipMalloc((void**) &_DEV_w_1,sizeof(int) *_SZ_w_1);
	hipMemcpy(_DEV_w_1,w,sizeof(int) *_SZ_w_1, hipMemcpyHostToDevice);
	int*_DEV_x_1;
	hipMalloc((void**) &_DEV_x_1,sizeof(int) *_SZ_x_1);
	hipMemcpy(_DEV_x_1,x,sizeof(int) *_SZ_x_1, hipMemcpyHostToDevice);
	int _NUM_THREADS_1= 20;
	float _NUM_BLOCKS_1=1;
	int _NUM_TILE_1=1;
	dim3 _THREADS_1(512);
	dim3 _BLOCKS_1(1);
	if(_NUM_THREADS_1 < _NTHREAD)
	{
		_THREADS_1.x=_NUM_THREADS_1;
	}
	else {
		 _THREADS_1.x=_NTHREAD;
		_NUM_BLOCKS_1=(_NUM_THREADS_1 % _NTHREAD == 0)?(_NUM_THREADS_1/_NTHREAD):((_NUM_THREADS_1/_NTHREAD)+1);
		if(_NUM_BLOCKS_1<_NBLOCK)
			_BLOCKS_1.x=_NUM_BLOCKS_1;
		else {
			_BLOCKS_1.x=_NBLOCK;
			int temp_1=_NUM_BLOCKS_1;
			_NUM_TILE_1=(temp_1 % _NBLOCK == 0)?(_NUM_BLOCKS_1/_NBLOCK):((_NUM_BLOCKS_1/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_1);
	#endif

	int _CUDA_TILE_1;
	for(i=0;i<=9;i+=17)
	for(_CUDA_TILE_1=0;_CUDA_TILE_1<_NUM_TILE_1;_CUDA_TILE_1++)
	{		_AFFINE_KERNEL_1<<<_BLOCKS_1,_THREADS_1>>>( _DEV_w_1, _SZ_w_1, _DEV_x_1, _SZ_x_1,1,i,0,9,_CUDA_TILE_1);
		hipDeviceSynchronize();
	}
	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_1);
	#endif
	hipMemcpy(w, _DEV_w_1,sizeof(int) *_SZ_w_1, hipMemcpyDeviceToHost);
	hipMemcpy(x, _DEV_x_1,sizeof(int) *_SZ_x_1, hipMemcpyDeviceToHost);
	hipFree(_DEV_w_1);
	hipFree(_DEV_x_1);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_1);
		pre_time_1 = (double) ((((&mid_start_1)->tv_sec * 1000000000) + (&mid_start_1)->tv_nsec) - (((&start_1)->tv_sec * 1000000000) + (&start_1)->tv_nsec)) / 1000000000;
		post_time_1 = (double) ((((&end_1)->tv_sec * 1000000000) + (&end_1)->tv_nsec) - (((&mid_end_1)->tv_sec * 1000000000) + (&mid_end_1)->tv_nsec)) / 1000000000;
		computeTime_1 = (double) ((((&mid_end_1)->tv_sec * 1000000000) + (&mid_end_1)->tv_nsec) - (((&mid_start_1)->tv_sec * 1000000000) + (&mid_start_1)->tv_nsec)) / 1000000000;
		runTime_1 = (double) ((((&end_1)->tv_sec * 1000000000) + (&end_1)->tv_nsec) - (((&start_1)->tv_sec * 1000000000) + (&start_1)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%%d,%%d,%%d,%%d,%%d,%%.14f,%%.14f,%%.14f,%%.14f,%%d\n",N,_NTHREAD*_NBLOCK,_THREADS_1.x,_BLOCKS_1.x,data,pre_time_1,computeTime_1,post_time_1,runTime_1,_CUDA_TILE_1);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%%f\n",runTime_1);
	#endif
	#endif
    // PARALLELIZE IN CUDA END
    
    return 0;
}




__global__ void _AFFINE_KERNEL_0(int* w,int  _SZ_w_0,int* x,int  _SZ_x_0,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
x[1+i] =(2 * i);
w[1+i] =(2 * i);
}}



__global__ void _AFFINE_KERNEL_1(int* w,int  _SZ_w_1,int* x,int  _SZ_x_1,int phi_count, int CUDA_i,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_i<=i)&&(i<(CUDA_i+17))&&(i<=CUDA_U_i)){
x[1+i-1] = -x[20-1-i];
w[1+i-1] = w[20-1-i];
}}
