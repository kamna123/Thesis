#include<stdio.h>
#include<stdlib.h>
#include<string.h>
/*cb gf*/
int main()
{
     int x[20],y[20],z[10],f[20];
   int i,a[50],k=5,j,b[300];
 
 
   //fasdf

int cd;
int jk;
int ty;
int kl;
#define _NTHREAD 512
#define _NBLOCK 65535
#include<hip/hip_runtime.h>

__global__ void _AFFINE_KERNEL_0(int* ,int ,int ,int ,int );

	int _SZ_a_0 = 50;
	int *_DEV_a_0;
	hipMalloc((void**) &_DEV_a_0,sizeof(int) *_SZ_a_0);
	hipMemcpy(_DEV_a_0,a,sizeof(int)*_SZ_a_0, hipMemcpyHostToDevice);
	int _NUM_THREADS_0= 50;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}
	int _CUDA_TILE_0;
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>( _DEV_a_0, _SZ_a_0,0,19, _CUDA_TILE_0);
		hipDeviceSynchronize();
	}
	hipMemcpy(a, _DEV_a_0,sizeof(int) *_SZ_a_0, hipMemcpyDeviceToHost);
	hipFree(_DEV_a_0);
#define _NTHREAD 512
#define _NBLOCK 65535


__global__ void _AFFINE_KERNEL_1(int* ,int ,int* ,int ,int* ,int ,int ,int ,int ,int ,int );

	int _SZ_a_1 = 50;
	int _SZ_b_1 = 300;
	int _SZ_z_1 = 10;
	int*_DEV_a_1;
	hipMalloc((void**) &_DEV_a_1,sizeof(int) *_SZ_a_1);
	hipMemcpy(_DEV_a_1,a,sizeof(int) *_SZ_a_1, hipMemcpyHostToDevice);
	int*_DEV_b_1;
	hipMalloc((void**) &_DEV_b_1,sizeof(int) *_SZ_b_1);
	hipMemcpy(_DEV_b_1,b,sizeof(int) *_SZ_b_1, hipMemcpyHostToDevice);
	int*_DEV_z_1;
	hipMalloc((void**) &_DEV_z_1,sizeof(int) *_SZ_z_1);
	hipMemcpy(_DEV_z_1,z,sizeof(int) *_SZ_z_1, hipMemcpyHostToDevice);
	int _NUM_THREADS_1= 50;
	float _NUM_BLOCKS_1=1;
	int _NUM_TILE_1=1;
	dim3 _THREADS_1(512);
	dim3 _BLOCKS_1(1);
	if(_NUM_THREADS_1 < _NTHREAD)
	{
		_THREADS_1.x=_NUM_THREADS_1;
	}
	else {
		 _THREADS_1.x=_NTHREAD;
		_NUM_BLOCKS_1=(_NUM_THREADS_1 % _NTHREAD == 0)?(_NUM_THREADS_1/_NTHREAD):((_NUM_THREADS_1/_NTHREAD)+1);
		if(_NUM_BLOCKS_1<_NBLOCK)
			_BLOCKS_1.x=_NUM_BLOCKS_1;
		else {
			_BLOCKS_1.x=_NBLOCK;
			int temp_1=_NUM_BLOCKS_1;
			_NUM_TILE_1=(temp_1 % _NBLOCK == 0)?(_NUM_BLOCKS_1/_NBLOCK):((_NUM_BLOCKS_1/_NBLOCK)+1);
		}
	}
	int _CUDA_TILE_1;
	for(i=0;i<=19;i+=1)
	for(_CUDA_TILE_1=0;_CUDA_TILE_1<_NUM_TILE_1;_CUDA_TILE_1++)
	{		_AFFINE_KERNEL_1<<<_BLOCKS_1,_THREADS_1>>>( _DEV_a_1, _SZ_a_1, _DEV_b_1, _SZ_b_1, _DEV_z_1, _SZ_z_1,1,i,0,19,_CUDA_TILE_1);
		hipDeviceSynchronize();
	}	hipMemcpy(a, _DEV_a_1,sizeof(int) *_SZ_a_1, hipMemcpyDeviceToHost);
	hipMemcpy(b, _DEV_b_1,sizeof(int) *_SZ_b_1, hipMemcpyDeviceToHost);
	hipMemcpy(z, _DEV_z_1,sizeof(int) *_SZ_z_1, hipMemcpyDeviceToHost);
	hipFree(_DEV_a_1);
	hipFree(_DEV_b_1);
	hipFree(_DEV_z_1);


    return 0;
}




__global__ void _AFFINE_KERNEL_0(int* a,int  _SZ_a_0,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
a[1+i] = 5;
}}



__global__ void _AFFINE_KERNEL_1(int* a,int  _SZ_a_1,int* b,int  _SZ_b_1,int* z,int  _SZ_z_1,int phi_count, int CUDA_i,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_i<=i)&&(i<(CUDA_i+1))&&(i<=CUDA_U_i)){
a[5+1+i] = b[1+1+i];
b[2+1+i] =((a[1+i+2] + 2) + z[2+1+i]);
}}

