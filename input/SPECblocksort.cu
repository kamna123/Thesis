#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
int main()
{
    int block[20],quadrant[20],i,j,k;
__global__ void _AFFINE_KERNEL_0(int* ,int ,int* ,int ,int ,int ,int );


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_0, end_0, mid_start_0, mid_end_0;
		double runTime_0, pre_time_0, post_time_0, computeTime_0;
	#endif
	int _SZ_block_0 = 20;
	int _SZ_quadrant_0 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_0);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_block_0;
	hipMalloc((void**) &_DEV_block_0,sizeof(int) *_SZ_block_0);
	hipMemcpy(_DEV_block_0,block,sizeof(int)*_SZ_block_0, hipMemcpyHostToDevice);
	int *_DEV_quadrant_0;
	hipMalloc((void**) &_DEV_quadrant_0,sizeof(int) *_SZ_quadrant_0);
	hipMemcpy(_DEV_quadrant_0,quadrant,sizeof(int)*_SZ_quadrant_0, hipMemcpyHostToDevice);
	int _NUM_THREADS_0= 20;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_0);
	#endif

	int _CUDA_TILE_0;
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>( _DEV_block_0, _SZ_block_0, _DEV_quadrant_0, _SZ_quadrant_0,0,19, _CUDA_TILE_0);
		hipDeviceSynchronize();
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_0);
	#endif
	hipMemcpy(block, _DEV_block_0,sizeof(int) *_SZ_block_0, hipMemcpyDeviceToHost);
	hipMemcpy(quadrant, _DEV_quadrant_0,sizeof(int) *_SZ_quadrant_0, hipMemcpyDeviceToHost);
	hipFree(_DEV_block_0);
	hipFree(_DEV_quadrant_0);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_0);

		pre_time_0 = (double) ((((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
		post_time_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec)) / 1000000000;
		computeTime_0 = (double) ((((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec) - (((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec)) / 1000000000;
		runTime_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%%d,%%d,%%d,%%d,%%d,%%.14f,%%.14f,%%.14f,%%.14f,%%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time_0,computeTime_0,post_time_0,runTime_0,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%%f\n",runTime_0);
	#endif
	#endif
    // PARALLELIZE IN CUDA START
__global__ void _AFFINE_KERNEL_1(int* ,int ,int* ,int ,int ,int ,int ,int ,int );


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_1, end_1, mid_start_1, mid_end_1;
		double runTime_1, pre_time_1, post_time_1, computeTime_1;
	#endif
	int _SZ_block_1 = 20;
	int _SZ_quadrant_1 = 20;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_1);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int*_DEV_block_1;
	hipMalloc((void**) &_DEV_block_1,sizeof(int) *_SZ_block_1);
	hipMemcpy(_DEV_block_1,block,sizeof(int) *_SZ_block_1, hipMemcpyHostToDevice);
	int*_DEV_quadrant_1;
	hipMalloc((void**) &_DEV_quadrant_1,sizeof(int) *_SZ_quadrant_1);
	hipMemcpy(_DEV_quadrant_1,quadrant,sizeof(int) *_SZ_quadrant_1, hipMemcpyHostToDevice);
	int _NUM_THREADS_1= 20;
	float _NUM_BLOCKS_1=1;
	int _NUM_TILE_1=1;
	dim3 _THREADS_1(512);
	dim3 _BLOCKS_1(1);
	if(_NUM_THREADS_1 < _NTHREAD)
	{
		_THREADS_1.x=_NUM_THREADS_1;
	}
	else {
		 _THREADS_1.x=_NTHREAD;
		_NUM_BLOCKS_1=(_NUM_THREADS_1 % _NTHREAD == 0)?(_NUM_THREADS_1/_NTHREAD):((_NUM_THREADS_1/_NTHREAD)+1);
		if(_NUM_BLOCKS_1<_NBLOCK)
			_BLOCKS_1.x=_NUM_BLOCKS_1;
		else {
			_BLOCKS_1.x=_NBLOCK;
			int temp_1=_NUM_BLOCKS_1;
			_NUM_TILE_1=(temp_1 % _NBLOCK == 0)?(_NUM_BLOCKS_1/_NBLOCK):((_NUM_BLOCKS_1/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_1);
	#endif

	int _CUDA_TILE_1;
	for(i=0;i<=19;i+=15)
	for(_CUDA_TILE_1=0;_CUDA_TILE_1<_NUM_TILE_1;_CUDA_TILE_1++)
	{		_AFFINE_KERNEL_1<<<_BLOCKS_1,_THREADS_1>>>( _DEV_block_1, _SZ_block_1, _DEV_quadrant_1, _SZ_quadrant_1,1,i,0,19,_CUDA_TILE_1);
		hipDeviceSynchronize();
	}
	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_1);
	#endif
	hipMemcpy(block, _DEV_block_1,sizeof(int) *_SZ_block_1, hipMemcpyDeviceToHost);
	hipMemcpy(quadrant, _DEV_quadrant_1,sizeof(int) *_SZ_quadrant_1, hipMemcpyDeviceToHost);
	hipFree(_DEV_block_1);
	hipFree(_DEV_quadrant_1);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_1);
		pre_time_1 = (double) ((((&mid_start_1)->tv_sec * 1000000000) + (&mid_start_1)->tv_nsec) - (((&start_1)->tv_sec * 1000000000) + (&start_1)->tv_nsec)) / 1000000000;
		post_time_1 = (double) ((((&end_1)->tv_sec * 1000000000) + (&end_1)->tv_nsec) - (((&mid_end_1)->tv_sec * 1000000000) + (&mid_end_1)->tv_nsec)) / 1000000000;
		computeTime_1 = (double) ((((&mid_end_1)->tv_sec * 1000000000) + (&mid_end_1)->tv_nsec) - (((&mid_start_1)->tv_sec * 1000000000) + (&mid_start_1)->tv_nsec)) / 1000000000;
		runTime_1 = (double) ((((&end_1)->tv_sec * 1000000000) + (&end_1)->tv_nsec) - (((&start_1)->tv_sec * 1000000000) + (&start_1)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%%d,%%d,%%d,%%d,%%d,%%.14f,%%.14f,%%.14f,%%.14f,%%d\n",N,_NTHREAD*_NBLOCK,_THREADS_1.x,_BLOCKS_1.x,data,pre_time_1,computeTime_1,post_time_1,runTime_1,_CUDA_TILE_1);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%%f\n",runTime_1);
	#endif
	#endif
    // PARALLELIZE IN CUDA END
}




__global__ void _AFFINE_KERNEL_0(int* block,int  _SZ_block_0,int* quadrant,int  _SZ_quadrant_0,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
block[1+i] =(2 *(i + 1));
quadrant[1+i] =(3 *(i + 1));
}}



__global__ void _AFFINE_KERNEL_1(int* block,int  _SZ_block_1,int* quadrant,int  _SZ_quadrant_1,int phi_count, int CUDA_i,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_i<=i)&&(i<(CUDA_i+15))&&(i<=CUDA_U_i)){
block[15+1+i] = block[1+i];
quadrant[15+1+i] = 0;
}}

