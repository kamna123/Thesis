#ifndef DATASET
	#define _NTHREAD 512
	#define _NBLOCK 65535
#endif

#include<hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
/*cb gf*/
int main()
{
     int x[20],y[20],z[10],f[20];
   int i,a[50],k=5,j,b[300];
 
 
   //fasdf

int cd;
int jk;
int ty;
int kl;
 /* for(i=1;i<=20;i=i+1)
    {
     a[2+2*i]=5;
    }*/
__global__ void _AFFINE_KERNEL_0(int* ,int ,int* ,int ,int* ,int ,int ,int ,int );

#define MIN(a,b) (((a)<(b))?(a):(b))


	#ifdef DATASET
		char* outfile = (char*)malloc(sizeof(char)*(strlen(readfile)+50));
		strcpy(outfile, readfile);
		strcat(outfile, ".data");
		FILE* fp;
		fp = fopen(outfile, "a");
	#endif

	#ifdef TIME
		struct timespec start_0, end_0, mid_start_0, mid_end_0;
		double runTime_0, pre_time_0, post_time_0, computeTime_0;
	#endif
	int _SZ_a_0 = 50;
	int _SZ_b_0 = 300;
	int _SZ_z_0 = 10;

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &start_0);
	#endif

	// ----------Allocating memory to Kernel Variable and copying them on device----------
	int *_DEV_a_0;
	hipMalloc((void**) &_DEV_a_0,sizeof(int) *_SZ_a_0);
	hipMemcpy(_DEV_a_0,a,sizeof(int) *_SZ_a_0, hipMemcpyHostToDevice);
	int *_DEV_b_0;
	hipMalloc((void**) &_DEV_b_0,sizeof(int) *_SZ_b_0);
	hipMemcpy(_DEV_b_0,b,sizeof(int) *_SZ_b_0, hipMemcpyHostToDevice);
	int *_DEV_z_0;
	hipMalloc((void**) &_DEV_z_0,sizeof(int) *_SZ_z_0);
	hipMemcpy(_DEV_z_0,z,sizeof(int) *_SZ_z_0, hipMemcpyHostToDevice);
	int _NUM_THREADS_0 = 50;
	float _NUM_BLOCKS_0=1;
	int _NUM_TILE_0=1;
	dim3 _THREADS_0(512);
	dim3 _BLOCKS_0(1);
	if(_NUM_THREADS_0 < _NTHREAD)
	{
		_THREADS_0.x=_NUM_THREADS_0;
	}
	else {
		 _THREADS_0.x=_NTHREAD;
		_NUM_BLOCKS_0=(_NUM_THREADS_0 % _NTHREAD == 0)?(_NUM_THREADS_0/_NTHREAD):((_NUM_THREADS_0/_NTHREAD)+1);
		if(_NUM_BLOCKS_0<_NBLOCK)
			_BLOCKS_0.x=_NUM_BLOCKS_0;
		else {
			_BLOCKS_0.x=_NBLOCK;
			int temp_0=_NUM_BLOCKS_0;
			_NUM_TILE_0=(temp_0 % _NBLOCK == 0)?(_NUM_BLOCKS_0/_NBLOCK):((_NUM_BLOCKS_0/_NBLOCK)+1);
		}
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_start_0);
	#endif

	int _CUDA_TILE_0;
	int ID_1_0=0, next_ID_1_0;
	while((ID_1_0<19))
	{
		next_ID_1_0 = MIN((((3)+(0)*ID_1_0)/(1)), (((2)+(1)*ID_1_0)/(1)));
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>( _DEV_a_0, _SZ_a_0, _DEV_b_0, _SZ_b_0, _DEV_z_0, _SZ_z_0,ID_1_0, MIN(next_ID_1_0,19), _CUDA_TILE_0);
			hipDeviceSynchronize();
		}
	for(_CUDA_TILE_0=0;_CUDA_TILE_0<_NUM_TILE_0;_CUDA_TILE_0++)
	{
		_AFFINE_KERNEL_0<<<_BLOCKS_0,_THREADS_0>>>( _DEV_a_0, _SZ_a_0, _DEV_b_0, _SZ_b_0, _DEV_z_0, _SZ_z_0,next_ID_1_0, 19, _CUDA_TILE_0);
			hipDeviceSynchronize();
		}
		ID_1_0=next_ID_1_0;
	}

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &mid_end_0);
	#endif
	hipMemcpy(a, _DEV_a_0,sizeof(int) *_SZ_a_0, hipMemcpyDeviceToHost);
	hipMemcpy(b, _DEV_b_0,sizeof(int) *_SZ_b_0, hipMemcpyDeviceToHost);
	hipMemcpy(z, _DEV_z_0,sizeof(int) *_SZ_z_0, hipMemcpyDeviceToHost);
	hipFree(_DEV_a_0);
	hipFree(_DEV_b_0);
	hipFree(_DEV_z_0);

	#ifdef TIME
		clock_gettime(CLOCK_MONOTONIC, &end_0);
		pre_time_0 = (double) ((((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
		post_time_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec)) / 1000000000;
		computeTime_0 = (double) ((((&mid_end_0)->tv_sec * 1000000000) + (&mid_end_0)->tv_nsec) - (((&mid_start_0)->tv_sec * 1000000000) + (&mid_start_0)->tv_nsec)) / 1000000000;
		runTime_0 = (double) ((((&end_0)->tv_sec * 1000000000) + (&end_0)->tv_nsec) - (((&start_0)->tv_sec * 1000000000) + (&start_0)->tv_nsec)) / 1000000000;
	#endif

	#ifdef DATASET
		fprintf(fp,"%%d,%%d,%%d,%%d,%%d,%%.14f,%%.14f,%%.14f,%%.14f,%%d\n",N,_NTHREAD*_NBLOCK,_THREADS_0.x,_BLOCKS_0.x,data,pre_time_0,computeTime_0,post_time_0,runTime_0,_CUDA_TILE_0);
		fclose(fp);
		fclose(f);
	#else
	#ifdef TIME
		printf("Runtime:%%f\n",runTime_0);
	#endif
	#endif


    return 0;
}




__global__ void _AFFINE_KERNEL_0(int* a,int  _SZ_a_0,int* b,int  _SZ_b_0,int* z,int  _SZ_z_0,int CUDA_L_i,int CUDA_U_i,int _CUDA_TILE)
{
	int i = gridDim.x*blockDim.x*_CUDA_TILE + blockDim.x*blockIdx.x + threadIdx.x;
	if((CUDA_L_i<=i)&&(i<=CUDA_U_i)){
a[5+1+i] = b[1+1+i];
b[2+2+2*i] =((a[1+i+2] + 2) + z[2+1+i]);
}}

